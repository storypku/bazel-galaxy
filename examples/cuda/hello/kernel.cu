#include "hip/hip_runtime.h"
#include <cstdio>

#include "hip/hip_runtime_api.h"
#include "examples/cuda/hello/kernel.h"

namespace {

__global__ void Hello() { std::printf("Hello World from GPU!\n"); }

void ReportIfError(hipError_t error) {
  if (error != hipSuccess) {
    std::fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
  }
}

}  // namespace

void HelloFromCuda() {
  Hello<<<1, 1>>>();

  ReportIfError(hipGetLastError());
  ReportIfError(hipDeviceSynchronize());
}
